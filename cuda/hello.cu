
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 7;
const int blocksize = 7;

/* Adds the an integer from the [b] array to a character in the same position
 * in the [a] array and stores the result back in [a]. Uses a multithreaded
 * pattern to add the two (each thread modifies a different index in parallel).
 * 
 * Requires: |a| = |b|.
 */
__global__
void hello(char* a, int* b) {
  a[threadIdx.x] += b[threadIdx.x];
}

/* Initializes the arrays and prints out the actual hello-world program.
 *
 * Requires: Computer supports CUDA hardware for multithreaded code to run.
 */
int main(int argc, char* argv[]) {
  char a[N] = "Hello ";
  int  b[N] = {47, 10, 6, 0, -11, 1, 0}; // Diffs between "Hello " and "world!"

  char* ad;
  int*  bd;
  const int csize = N * sizeof(char);
  const int isize = N * sizeof(int);

  printf("%s", a);   // Print out "Hello "

  hipMalloc((void**) &ad, csize);
  hipMalloc((void**) &bd, isize);
  
  hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
  hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);                   // Set up blocks to copy.
  hello<<<dimGrid, dimBlock>>>(ad, bd); // Works in theory, but requires CUDA.
  hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);

  hipFree(ad);
  hipFree(bd);

  printf("%s\n", a); // Print out "world!"
  return EXIT_SUCCESS;
}
